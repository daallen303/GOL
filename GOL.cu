
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;


struct Cell{
	char status;
	int count;
};


/*
char getStatus()
{
	return status;
}
int getCount()
{
	return count;
}
void setCount(int num)
{
		count = num;
}

void initStatus(char c)
{
	status = c;
}

*/
/*
char setStatus(int num, char status) //Each line ends with newline character \n (Unix formatting)
{
		char newStat = status;
		if(status == 'X') //check if it's alive
		{
		if(num < 2) newStat = '-';//dead less than 2 living neighbours
		else if(num <= 3) cout << "ok"; //do nothing status is already alive
		else newStat = '-';//dead greater than 3 living neighbours
		}
		else{
			if(num == 3) newStat = 'X'; // dead to alive
		}
		return newStat;
}


void checkAdjCells(int rows, int cols, int cIndex, Cell A[])
	{
		int i, j, iIndex, jIndex, count = 0;
		iIndex = cIndex/cols; //row index
		jIndex = cIndex%cols; // col index
		for(i = iIndex-1; i <= iIndex+1; i++)
		{
			for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
			{
				// i<0 can't have negative index
				//i>rows j > cols can't have index larger than array Max
				if(i>=0 && i<=rows && j >=0 && j<= cols && A[i*cols+j].status == 'X' && (i*cols+j!= cIndex)) count++;
			}
		}
		A[cIndex].count = count;
	}
*/
__global__
void callCheck(int rows, int cols,Cell A[])
{
	int i, k, j, count, iIndex, jIndex;
	char newStat;
    int index = blockIdx.x * blockDim.x + threadIdx.x; //index of current thread
	//int stride = blockDim.x *gridDim.x; total threads grid striping
	for(i = index; i < cols*rows; i++) 
			{
				//checkAdjCells(rows,cols, k, A);
				iIndex = i/cols; //row index
				jIndex = i%cols; // col index
				count = 0;
				for(k = iIndex-1; k <= iIndex+1; k++)
				{
					for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
					{
						// i<0 can't have negative index
						//i>rows j > cols can't have index larger than array Max
						if(k>=0 && k<=rows && j >=0 && j<= cols && A[i*cols+j].status == 'X' && (k*cols+j!= i)) count++;
					}
				}
				A[i].count = count;
				newStat = A[i].status;
						if(newStat == 'X') //check if it's alive
						{
						if(count < 2) newStat = '-';//dead less than 2 living neighbours
						else if(count <= 3) newStat = 'X'; //do nothing status is already alive
						else newStat = '-';//dead greater than 3 living neighbours
						}
						else{
							if(count == 3) newStat = 'X'; // dead to alive
						}
				A[i].status = newStat;
			}
}	


int main()
{
	int rows = 10;
	int cols = 10;
	int i,j, cIndex;
	char temp;
	
	Cell *A;
	hipMallocManaged(&A, rows*cols*(8)); //allocates bytes from device heap and returns pointer to allocated memory or null
	ifstream fin;
	fin.open("./input.txt");
	
	for(i = 0; i < rows; i++)
		for(j = 0; j<cols; j++)
		{   
			fin >> temp;
			cIndex = i*cols+j;
			A[cIndex].status = temp;
			A[cIndex].count = 0;
			//cout << A[i *cols + j].getStatus();
		}
	
	callCheck<<<1,1>>>(rows, cols, A);
	hipDeviceSynchronize();
	hipFree(A);
				//	A[23].setCount(checkAdjCells(rows,cols,23, A));
				//	cout << "Status " << A[23].getStatus() << " Count" << A[23].getCount();
	fin.close();
cin.get();
	//cudaMallocManaged(sizeof(char)*rows*cols);
	//cudaMemcpy(hostToDevice)

}
