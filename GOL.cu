
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;


/*struct Cell{
	char status;
	int count;
};

*/
/*
char getStatus()
{
	return status;
}
int getCount()
{
	return count;
}
void setCount(int num)
{
		count = num;
}

void initStatus(char c)
{
	status = c;
}

*/
/*
char setStatus(int num, char status) //Each line ends with newline character \n (Unix formatting)
{
		char newStat = status;
		if(status == 'X') //check if it's alive
		{
		if(num < 2) newStat = '-';//dead less than 2 living neighbours
		else if(num <= 3) cout << "ok"; //do nothing status is already alive
		else newStat = '-';//dead greater than 3 living neighbours
		}
		else{
			if(num == 3) newStat = 'X'; // dead to alive
		}
		return newStat;
}


void checkAdjCells(int rows, int cols, int cIndex, Cell A[])
	{
		int i, j, iIndex, jIndex, count = 0;
		iIndex = cIndex/cols; //row index
		jIndex = cIndex%cols; // col index
		for(i = iIndex-1; i <= iIndex+1; i++)
		{
			for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
			{
				// i<0 can't have negative index
				//i>rows j > cols can't have index larger than array Max
				if(i>=0 && i<=rows && j >=0 && j<= cols && A[i*cols+j].status == 'X' && (i*cols+j!= cIndex)) count++;
			}
		}
		A[cIndex].count = count;
	}
*/
__global__
void callCheck(int rows, int cols,char A[], int B[])
{
	int i, k, j, count, iIndex, jIndex;
	char newStat;
    int index = blockIdx.x * blockDim.x + threadIdx.x; //index of current thread
	//int stride = blockDim.x *gridDim.x; total threads grid striping
	for(i = index; i < cols*rows; i++) 
			{
				//checkAdjCells(rows,cols, k, A);
				iIndex = i/cols; //row index
				jIndex = i%cols; // col index
				count = 0;
				for(k = iIndex-1; k <= iIndex+1; k++)
				{
					for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
					{
						// i<0 can't have negative index
						//i>rows j > cols can't have index larger than array Max
						if(k>=0 && k<=rows && j >=0 && j<= cols && A[k*cols+j] == 'X' && (k*cols+j!= i)) count++;
					}
				}
				B[i] = count;
				newStat = A[i];
						if(newStat == 'X') //check if it's alive
						{
						if(count < 2) newStat = '-';//dead less than 2 living neighbours
						else if(count <= 3) newStat = 'X'; //do nothing status is already alive
						else newStat = '-';//dead greater than 3 living neighbours
						}
						else{
							if(count == 3) newStat = 'X'; // dead to alive
						}
				A[i] = newStat;
			}
}	

int main()
{
	int rows = 10;
	int cols = 10;
	int i,j, cIndex;
	char temp;
	
	// two sepreate array coalesced reads cuda
	char S[rows*cols];
	int C[rows*cols];
	ifstream fin;
	fin.open("./input.txt");
	
	for(i = 0; i < rows; i++)
		for(j = 0; j<cols; j++)
		{   
			fin >> temp;
			cIndex = i*cols+j;
			S[cIndex]= temp;
			C[cIndex] = 0;
			cout << i*cols+j << " index " << S[cIndex] << " = status " << C[cIndex] << " = count" << endl;
			//cout << A[i *cols + j].getStatus();
		}
	fin.close();
	char *A;
	int *B;
	hipMalloc(&A, rows*cols*(sizeof(int)));
	hipMalloc(&B, rows*cols*(sizeof(int)));//allocates bytes from device heap and returns pointer to allocated memory or null
	for(i = 0; i < rows; i++)
	{
			for(j = 0; j<cols; j++)
			{   
				hipMemcpy(&A[i*cols+j], &S[i*cols+j], sizeof(char), hipMemcpyHostToDevice);
				hipMemcpy(&B[i*cols+j], &C[i*cols+j], sizeof(int), hipMemcpyHostToDevice);
			}

				//	A[23].setCount(checkAdjCells(rows,cols,23, A));
				//	cout << "Status " << A[23].getStatus() << " Count" << A[23].getCount();
	}
	int l = 0;
	while(l< 10){
		        //     <<<number of blocks, number of threads per block>>>
	callCheck<<<1,100>>>(rows, cols, A, B);
	hipDeviceSynchronize();
	for(i = 0; i < rows; i++)
		{
				
				for(j = 0; j<cols; j++)
				{   
					
					hipMemcpy(&S[i*cols+j], &A[i*cols+j], sizeof(char), hipMemcpyDeviceToHost);
					hipMemcpy(&C[i*cols+j], &B[i*cols+j], sizeof(int), hipMemcpyDeviceToHost);
					//cout << i*cols+j << " index " << S[i*cols+j] << " status " << C[i*cols+j] << " count " << endl;
					cout << S[i*cols+j] << " ";
				}
				cout << endl;
					//	A[23].setCount(checkAdjCells(rows,cols,23, A));
					//	cout << "Status " << A[23].getStatus() << " Count" << A[23].getCount();
		}
	l++;
	}
	hipFree(A);
	hipFree(B);
	
	cin.get();
	//cudaMallocManaged(sizeof(char)*rows*cols);
	//cudaMemcpy(hostToDevice)

}
