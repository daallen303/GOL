
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <stdlib.h>
#include <string>

using namespace std;

const int THREADS_PER_BLOCK = 512;

__global__
void callCheck(int rows, int cols,char A[])
{
	int i, k, j, count, iIndex, jIndex;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    int rowIndex, colIndex;//index of current thread
	iIndex = i/cols; //row index
	jIndex = i%cols; // col index
	count = 0;
	for(k = iIndex-1; k <= iIndex+1; k++)
	{
		for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
		{
			// k<0 j >0 can't have negative index
			//k>rows j > cols can't have index larger than array Max
			if(k<0) rowIndex = rows-1;
			else if(k>=rows)rowIndex = 0;
			else rowIndex = k;
			if(j<0) colIndex = cols-1;
			else if(j>=cols) colIndex = 0;
			else colIndex = j;
			if (A[rowIndex*cols+colIndex] == 'X' && (rowIndex*cols+colIndex!= i)) count++;
		}
	}
	if(A[i] == 'X') //check if it's alive
	{
	if(count < 2) A[i] = '-';//dead less than 2 living neighbours
	else if(count <= 3) A[i] = 'X'; //do nothing status is already alive
	else A[i] = '-';//dead greater than 3 living neighbours
	}else{ //dead cell
			if(count == 3) A[i] = 'X';// dead to alive
	}
}	

int main(int argc, char *argv[])
{
	int i,j, rows, cols;
	char temp = '=';
	rows = 1;
	cols = 1;
	vector<char> tempS;
	ifstream fin;
	ofstream fout;
	bool printAll = false;
	int opts = 0;
	string input;
	int iterations = 1;
	
	while(opts < argc)
	{
		if(string(argv[opts]) == "-i") iterations = strtol(argv[opts+1], NULL, 10);
		if(string(argv[opts]) == "-v") printAll = true;
		if(opts == argc-1)
		{
			string ext;
			string temp = argv[opts];
			for(i = temp.length()-4; i < temp.length(); i++) ext += temp[i];
			if(ext == ".txt") input = temp;
		}
		opts++;
	}
	
	fin.open(input.c_str());
	if(fin){
	fout.open("output.txt");
	i=0;
	fin >> temp;
	int totalcount = 0; //total number of elements
	while(!fin.eof())
	{
		totalcount++;
		if(temp == 'X' || temp == '-')
		{
			if(fin.peek() == '\n')
				{
				   rows++;
				}else if(rows == 1)cols++;
		tempS.push_back(temp); //read in status 
		}else cout << "Invalid input = " << temp << endl;
		fin >> temp;
		i++;
	}
	fin.close();
	
	int Array_size = cols*rows;
	if(Array_size >8){
	if(totalcount== rows*cols){
	char S[rows*cols];
	for(j=0; j<rows*cols; j++)
	{
		S[j]= tempS[j];
	}
	
	tempS.clear();
	
	fout << "Initial step" << endl;
	for(i = 0; i < rows; i++)
		{
					
			for(j = 0; j<cols; j++)
			{   
				fout << S[i*cols+j];
			}
			fout << endl;
		}
	fout << endl;
	fout << endl;
	
	char *A;
	int GD;
	i=1;
	while(i <= THREADS_PER_BLOCK)
		{
		   if (Array_size%i == 0) GD = i;//find greatest denominator of Array_size < THREADS_PER_BLOCK
		   i++;
		}
	hipMalloc((void** ) &A, rows*cols*(sizeof(char)));	//allocates bytes from device heap and returns pointer to allocated memory or null
	hipMemcpy(A, S, rows*cols*sizeof(char), hipMemcpyHostToDevice);
	//cout << Array_size%(Array_size/(THREADS_PER_BLOCK-(Array_size%THREADS_PER_BLOCK))) << endl;
	//cout << THREADS_PER_BLOCK%(Array_size/(THREADS_PER_BLOCK-(Array_size%THREADS_PER_BLOCK))) << endl;
	int l = 0;
	while(l < iterations){
    //     <<<number of blocks, number of threads per block>>>
	callCheck<<<Array_size/GD,GD>>>(rows,cols,A);
	//callCheck<<<(Array_size+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(rows,cols,A); I get memcheck errors using this
	hipDeviceSynchronize();
	
	if(printAll == true || l == iterations-1)
	{
		hipMemcpy(S, A, rows*cols*sizeof(char), hipMemcpyDeviceToHost);
		fout << "Step " << l+1 << endl;
	for(i = 0; i < rows; i++)
		{
				
			for(j = 0; j<cols; j++)
			{   
				fout << S[i*cols+j];
			}
			fout << endl;
		}
	    fout << endl;
	    fout << endl;
	}
	l++;
	}
	hipFree(A);
	fout.close();
	cout << "All Done";
		}else cout << "Matrix is not even";
	}else cout <<"Matrix must be at least 9 elements";
	}else cout<< "Could not find the input file please try running again with valid file";
	cin.get();

}

