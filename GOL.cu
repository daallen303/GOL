
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <stdlib.h>
#include <string>

using namespace std;

const int THREADS_PER_BLOCK = 512;

__global__
void callCheck(int rows, int cols,char A[], int B[])
{
	int i, k, j, count, iIndex, jIndex;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    int rowIndex, colIndex;//index of current thread
	//int stride = blockDim.x *gridDim.x; total threads grid striping 
				//checkAdjCells(rows,cols, k, A);
	iIndex = i/cols; //row index
	jIndex = i%cols; // col index
	count = 0;
	for(k = iIndex-1; k <= iIndex+1; k++)
	{
		for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
		{
			// k<0 j >0 can't have negative index
			//k>rows j > cols can't have index larger than array Max
			if(k<0) rowIndex = rows-1;
			else if(k>=rows)rowIndex = 0;
			else rowIndex = k;
			if(j<0) colIndex = cols-1;
			else if(j>=cols) colIndex = 0;
			else colIndex = j;
			if (A[rowIndex*cols+colIndex] == 'X' && (rowIndex*cols+colIndex!= i)) count++;
		}
	}
	B[i] = count;
	if(A[i] == 'X') //check if it's alive
	{
	if(B[i] < 2) A[i] = '-';//dead less than 2 living neighbours
	else if(B[i] <= 3) A[i] = 'X'; //do nothing status is already alive
	else A[i] = '-';//dead greater than 3 living neighbours
	}
	else{ //dead cell
			if(B[i] == 3) A[i] = 'X';// dead to alive
	}
}	

int main(int argc, char *argv[])
{
	int i,j, rows, cols;
	char temp = '=';
	rows = 1;
	cols = 1;
	// two sepreate array coalesced reads cudachar S[rows*cols];
	vector<char> tempS;
	ifstream fin;
	ofstream fout;
	bool printAll = false;
		int opts = 0;
		string input;
		int iterations = 1;
		while(opts < argc)
		{
			if(string(argv[opts]) == "-i") iterations = strtol(argv[opts+1], NULL, 10);
			if(string(argv[opts]) == "-v") printAll = true;
			if(opts == argc-1){
				string ext;
				string temp = argv[opts];
				for(i = temp.length()-4; i < temp.length(); i++) ext += temp[i];
				if(ext == ".txt") input = temp;
			}
			opts++;
		}
	fin.open(input.c_str());
	if(fin){
	fout.open("output.txt");
	i=0;
	fin >> temp;
	int totalcount = 0; //total number of elements
	while(!fin.eof())
	{
		totalcount++;
		if(temp == 'X' || temp == '-')
		{
			if(fin.peek() == '\n')
				{
				rows++;
				}
			else if(rows == 1)cols++;
			tempS.push_back(temp); //read in status 
		}else cout << "Invalid input = " << temp << endl;
		fin >> temp;
		i++;
	}
	fin.close();
	if(cols*rows >8){
		if(totalcount== rows*cols){
	int C[rows*cols];
	char S[rows*cols];
	for(j=0; j<rows*cols; j++)
	{
		C[j]=-1;
		S[j]= tempS[j];
		
	}
	
	tempS.clear();
	
	fout << "Initial step" << endl;
	for(i = 0; i < rows; i++)
			{
					
					for(j = 0; j<cols; j++)
					{   
						
						
						fout << S[i*cols+j];
					}
					fout << endl;
			}
	fout << endl;
		fout << endl;
	char *A;
	int *B;
	hipMalloc((void** ) &A, rows*cols*(sizeof(char)));
	hipMalloc((void** ) &B, rows*cols*(sizeof(int)));//allocates bytes from device heap and returns pointer to allocated memory or null
	hipMemcpy(A, S, rows*cols*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(B, C, rows*cols*sizeof(int), hipMemcpyHostToDevice);
	
	int l = 0;
	while(l < iterations){
 //     <<<number of blocks, number of threads per block>>>
	if(rows*cols < THREADS_PER_BLOCK)callCheck<<<1,rows*cols>>>(rows, cols, A, B); // one block of rows*cols threads
	else callCheck<<<cols*rows/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(rows,cols,A,B);
	hipDeviceSynchronize();
	hipMemcpy(S, A, rows*cols*sizeof(char), hipMemcpyDeviceToHost);
	hipMemcpy(C, B, rows*cols*sizeof(int), hipMemcpyDeviceToHost);
	if(printAll == true || l == iterations-1)
	{
		fout << "Step " << l+1 << endl;
	for(i = 0; i < rows; i++)
		{
				
				for(j = 0; j<cols; j++)
				{   
					fout << S[i*cols+j];
				}
				fout << endl;
		}
	fout << endl;
	fout << endl;
	}
	l++;
	}
	hipFree(A);
	hipFree(B);
	fout.close();
	cout << "All Done";
		}else cout << "Matrix is not even";
	}else cout <<"Matrix must be at least 9 elements";
	}else cout<< "Could not find the input file please try running again with valid file";
	cin.get();

}

