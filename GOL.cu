
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>

using namespace std;

const int THREADS_PER_BLOCK = 512;

__global__
void callCheck(int rows, int cols,char A[], int B[])
{
	int i, k, j, count, iIndex, jIndex;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    int rowIndex, colIndex;//index of current thread
	//int stride = blockDim.x *gridDim.x; total threads grid striping 
				//checkAdjCells(rows,cols, k, A);
				iIndex = i/cols; //row index
				jIndex = i%cols; // col index
				count = 0;
				for(k = iIndex-1; k <= iIndex+1; k++)
				{
					for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
					{
						// i<0 can't have negative index
						//i>rows j > cols can't have index larger than array Max
						if(k<0) rowIndex = rows;
						else if(k>rows)rowIndex = 0;
						else rowIndex = k;
						if(j<0) colIndex = cols;
						else if(j>cols) colIndex = 0;
						else colIndex = j;
						if (A[rowIndex*cols+colIndex] == 'X' && (rowIndex*cols+colIndex!= i)) count++;
					}
				}
									B[i] = count;
										if(A[i] == 'X') //check if it's alive
										{
										if(B[i] < 2) A[i] = '-';//dead less than 2 living neighbours
										else if(B[i] <= 3) A[i] = 'X'; //do nothing status is already alive
										else A[i] = '-';//dead greater than 3 living neighbours
										}
										else{ //dead cell
											if(B[i] == 3) A[i] = 'X';// dead to alive
										}
}	

int main()
{
	int i,j, rows, cols;
	char temp;
	rows = 1;
	cols = 1;
	// two sepreate array coalesced reads cuda
	vector <char> S;
	vector <int> C;
	ifstream fin;
	ofstream fout;
	fin.open("./input.txt");
	fout.open("output.txt");
	fin >> temp;
	while(!fin.eof())
	{
		if(fin.peek() == '\n')rows++;
		else if(rows == 1)cols++;
		if(temp == 'X' || temp == '-')
		{
			S.push_back(temp); //read in status
			C.push_back(-1); //initialize count of Adjcells
		}else cout << "Invalid input = " << temp << endl;
		fin >> temp;
	}
	
	char *A;
	int *B;
	hipMalloc(&A, rows*cols*(sizeof(char)));
	hipMalloc(&B, rows*cols*(sizeof(int)));//allocates bytes from device heap and returns pointer to allocated memory or null
	for(i = 0; i < rows; i++)
	{
			for(j = 0; j<cols; j++)
			{   
				hipMemcpy(&A[i*cols+j], &S[i*cols+j], sizeof(char), hipMemcpyHostToDevice);
				hipMemcpy(&B[i*cols+j], &C[i*cols+j], sizeof(int), hipMemcpyHostToDevice);
			}
	}
	int l = 0;
	while(l< 5){
 //     <<<number of blocks, number of threads per block>>>
	callCheck<<<cols*rows/THREADS_PER_BLOCK+1,THREADS_PER_BLOCK>>>(rows, cols, A, B); // one block of rows*cols threads
	hipDeviceSynchronize();
	for(i = 0; i < rows; i++)
		{
				
				for(j = 0; j<cols; j++)
				{   
					
					hipMemcpy(&S[i*cols+j], &A[i*cols+j], sizeof(char), hipMemcpyDeviceToHost);
					hipMemcpy(&C[i*cols+j], &B[i*cols+j], sizeof(int), hipMemcpyDeviceToHost);
					fout << S[i*cols+j];
				}
				fout << endl;
		}
	fout << endl;
	fout << endl;
	l++;
	}
	hipFree(A);
	hipFree(B);
	cout << "All Done";
	cin.get();

}


/*struct Cell{
	char status;
	int count;
};


int checkAdjCells(int rows, int cols, int cIndex, char A[])
	{
		int i, j, iIndex, jIndex, count = 0;
		iIndex = cIndex/cols; //row index
		jIndex = cIndex%cols; // col index
		for(i = iIndex-1; i <= iIndex+1; i++)
		{
			for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
			{
				// i<0 can't have negative index
				//i>rows j > cols can't have index larger than array Max
				if(i>=0 && i<=rows && j >=0 && j<= cols && A[i*cols+j] == 'X' && (i*cols+j!= cIndex)) count++;
			}
		}
		return count;
	}

char setStatus(int num, char status) //Each line ends with newline character \n (Unix formatting)
{
		//cout << num << " ";
		char newStat = status;
		if(status == 'X') //check if it's alive
		{
		if(num < 2) newStat = '-';//dead less than 2 living neighbours
		else if(num <= 3) newStat = 'X'; //do nothing status is already alive
		else newStat = '-';//dead greater than 3 living neighbours
		}
		else{
			if(num == 3) newStat = 'X'; // dead to alive
		}
		return newStat;
}

char getStatus()
{
	return status;
}
int getCount()
{
	return count;
}
void setCount(int num)
{8
		count = num;
}

void initStatus(char c)
{
	status = c;
}

*/
/*
char setStatus(int num, char status) //Each line ends with newline character \n (Unix formatting)
{
		char newStat = status;
		if(status == 'X') //check if it's alive
		{
		if(num < 2) newStat = '-';//dead less than 2 living neighbours
		else if(num <= 3) cout << "ok"; //do nothing status is already alive
		else newStat = '-';//dead greater than 3 living neighbours
		}
		else{
			if(num == 3) newStat = 'X'; // dead to alive
		}
		return newStat;
}


void checkAdjCells(int rows, int cols, int cIndex, Cell A[])
	{
		int i, j, iIndex, jIndex, count = 0;
		iIndex = cIndex/cols; //row index
		jIndex = cIndex%cols; // col index
		for(i = iIndex-1; i <= iIndex+1; i++)
		{
			for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
			{
				// i<0 can't have negative index
				//i>rows j > cols can't have index larger than array Max
				if(i>=0 && i<=rows && j >=0 && j<= cols && A[i*cols+j].status == 'X' && (i*cols+j!= cIndex)) count++;
			}
		}
		A[cIndex].count = count;
	}

for(int m = 0; m < 5; m++){
	for(i = 0; i < rows; i++)
		{
			for(j = 0; j<cols; j++)
			{   
				cIndex = i*cols+j;
				C[cIndex]= checkAdjCells(rows,cols,cIndex, S);
				cout << S[cIndex] << C[cIndex];
				//cout << A[i *cols + j].getStatus();
			}
			cout << endl;
		}
	cout << endl;
	for(i = 0; i < rows; i++)
			{
				for(j = 0; j<cols; j++)
				{   
					cIndex = i*cols+j;
					S[cIndex] = setStatus(C[cIndex], S[cIndex]);
					//cout << A[i *cols + j].getStatus();
					cout << S[cIndex] << C[cIndex];
				}
				cout << endl;
			}
}
	
	cout << endl;
	cout << endl;
	fin.close();
*/
