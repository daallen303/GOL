#include <iostream>
#include <fstream>

using namespace std;


class Cell{
	private:
	char status;
	int count;

	public:

	void setCount(int num)
	{
		count = num;
	}
	void initStatus(char c)
	{
		status = c;
	}
	void setStatus(int num) //Each line ends with newline character \n (Unix formatting)
	{
		if(status == 'X') //check if it's alive
		{
		if(num < 2) status = '-';//dead less than 2 living neighbours
		else if(num <= 3) cout << "ok"; //do nothing status is already alive
		else status = '-';//dead greater than 3 living neighbours
		}
		else{
			if(num == 3) status = 'X'; // dead to alive
		}
	}
	char getStatus()
	{
	   return status;
	}
	int getCount()
	{
		return count;
	}


};

int checkAdjCells(int rows, int cols, int cIndex, Cell A[])
	{
	cout <<cIndex << endl;
		int i, j, iIndex, jIndex, count = 0;git@git.aetheris.co:daniel.allen/Cuda_GOL.git
		iIndex = cIndex/cols; //row index
		jIndex = cIndex%cols; // col index
		for(i = iIndex-1; i <= iIndex+1; i++)
		{
			cout << "i = " << i << endl;
			for (j = jIndex-1; j <= jIndex+1; j++)
			{
				cout << "j = " << j << endl;
				// i<0 can't have negative index
				//i>rows j > cols can't have index larger than array Max
				if(i>=0 && i<=rows && j >=0 && j<= cols && A[i*cols+j].getStatus() == 'X' && (i*cols+j!= cIndex)) count++;
			}
		}
		return count;
	}


int main()
{
	int rows = 10;
	int cols = 10;
	int i,j, cIndex;
	char temp;
	Cell A[cols*rows];

	ifstream fin;
	fin.open("./input.txt");
	for(i = 0; i < rows; i++)
		for(j = 0; j<cols; j++)
		{
			fin >> temp;
			cIndex = i*cols+j;
			A[cIndex].initStatus(temp);
			A[cIndex].setCount(0);
			cout << A[i *cols + j].getStatus();
		}
					A[23].setCount(checkAdjCells(rows,cols,23, A));
					cout << "Status " << A[23].getStatus() << " Count" << A[23].getCount();
	fin.close();
cin.get();
	//hipMallocManaged(sizeof(char)*rows*cols);
	//hipMemcpy(hostToDevice)

}
