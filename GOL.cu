
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <stdlib.h>
#include <string>
#include <unistd.h>

using namespace std;

const int THREADS_PER_BLOCK = 512;

__global__ 
void setStatus(char A[], int B[])
{
	int i;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	if(A[i] == 'X') //check if it's alive
	{
		if(B[i] < 2)
		{
			A[i] = '-';//dead less than 2 living neighbours
		}
		else if(B[i] <= 3)
		{
			A[i] = 'X'; //do nothing status is already alive
		}
		else 
		{
			A[i] = '-';//dead greater than 3 living neighbours
		}
	}
	else
	{ //dead cell
		if(B[i] == 3)
		{
			A[i] = 'X';// dead to alive
		}
	}
}

__global__
void callCheck(int rows, int cols,char A[], int B[])
{
	int i, k, j, count, iIndex, jIndex;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	int rowIndex, colIndex;//index of current thread
	iIndex = i/cols; //row index
	jIndex = i%cols; // col index
	count = 0;
	
	for(k = iIndex-1; k <= iIndex+1; k++)
	{
	    for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
		{
			// k < 0 j >0 can't have negative index
			//k > rows j > cols can't have index larger than array Max
	    	
			if(k<0)
			{
				rowIndex = rows-1;
			}
			else if(k>=rows)
			{
				rowIndex = 0;
			}
			else
			{
				rowIndex = k;
			}
			if(j<0)
			{
				colIndex = cols-1;
			}
			else if(j>=cols)
			{
				colIndex = 0;
			}
			else
			{
				colIndex = j;
			}
			if (A[rowIndex*cols+colIndex] == 'X' && (rowIndex*cols+colIndex!= i))
			{
				count++;
			}
		}
	}
	B[i] = count;
	
}


	

int main(int argc, char *argv[])
{
	int i,j, rows, cols;
	char temp = '=';
	rows = 1;
	cols = 1;
	vector<char> tempS;
	ifstream fin;
	ofstream fout;
	bool printAll = false;
	int opts = 0;
	string input;
	int iterations = 1;
	
	while(opts < argc)
	{
		if(string(argv[opts]) == "-i")
		{
			iterations = strtol(argv[opts+1], NULL, 10);
		}
		if(string(argv[opts]) == "-v")
		{
			printAll = true;
		}
		if(opts == argc-1)
		{
			string ext;
			string temp = argv[opts];
			for(i = temp.length()-4; i < temp.length(); i++) ext += temp[i];
			if(ext == ".txt") input = temp;
		}
		opts++;
	}
	
	fin.open(input.c_str());
	if(!fin)
	{
		cout<< "Could not find the input file please try running again with valid file";
		exit(1);
	}
	fout.open("output.txt");
	i=0;
	fin >> temp;
	int totalcount = 0;   //total number of elements
	
	while(!fin.eof())
	{
		totalcount++;
		if(temp == 'X' || temp == '-')
		{
			if(fin.peek() == '\n')
			{
				rows++;
			}else if(rows == 1)
			{
				cols++;
			}
		tempS.push_back(temp); //read in status 
		}
		else
		{
			cout << "Invalid input = " << temp << endl;
		}
		fin >> temp;
		i++;
	}
	fin.close();
	
	int Array_size = cols*rows;
	
	if(Array_size <= 8)
	{
		cout <<"Matrix must be at least 9 elements";
		exit(1);
	}
	if(totalcount != Array_size)
	{
		cout << "Matrix is not even";
		exit(1);
	}
	char S[Array_size];
	
	for(j=0; j<Array_size; j++)
	{
		S[j]= tempS[j];
	}
	
	tempS.clear();

	char *A;
	int *B;
	int GD;
	i=1;
	
	while(i <= THREADS_PER_BLOCK)
	{
		if (Array_size%i == 0)
			{
			GD = i;	//find greatest denominator of Array_size < THREADS_PER_BLOCK
			}
		   i++;
	}
	
	hipMalloc((void** ) &A, Array_size*(sizeof(char)));
	hipMalloc((void** ) &B, Array_size*(sizeof(int)));	//allocates bytes from device heap and returns pointer to allocated memory or null
	hipMemcpy(A, S, Array_size*sizeof(char), hipMemcpyHostToDevice);
	
	int l = 0;
	while(l < iterations)
	{

		callCheck<<<Array_size/GD,GD>>>(rows,cols,A, B);
		setStatus<<<Array_size/GD,GD>>>(A, B);
	
	
		if(printAll == true || l == iterations-1)
		{
			hipDeviceSynchronize();
			hipMemcpy(S, A, Array_size*sizeof(char), hipMemcpyDeviceToHost);
			printf("\033[2J\033[H");
			for(i = 0; i < rows; i++)
			{
				
				for(j = 0; j<cols; j++)
				{   
					cout << S[i*cols+j];
				}
				cout << endl;
			}
		}
		l++;
	}
	hipFree(A);
	hipFree(B);
	
	return 0;

}

