
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;


__global__
void callCheck(int rows, int cols,char A[], int B[])
{
	int i, k, j, count, iIndex, jIndex;
    i = blockIdx.x * blockDim.x + threadIdx.x; //index of current thread
	//int stride = blockDim.x *gridDim.x; total threads grid striping 
				//checkAdjCells(rows,cols, k, A);
				iIndex = i/cols; //row index
				jIndex = i%cols; // col index
				count = 0;
				for(k = iIndex-1; k <= iIndex+1; k++)
				{
					for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
					{
						// i<0 can't have negative index
						//i>rows j > cols can't have index larger than array Max
						if(k>=0 && k<=rows && j >=0 && j<= cols && A[k*cols+j] == 'X' && (k*cols+j!= i)) count++;
					}
				}
									B[i] = count;
										if(A[i] == 'X') //check if it's alive
										{
										if(B[i] < 2) A[i] = '-';//dead less than 2 living neighbours
										else if(B[i] <= 3) A[i] = 'X'; //do nothing status is already alive
										else A[i] = '-';//dead greater than 3 living neighbours
										}
										else{ //dead cell
											if(B[i] == 3) A[i] = 'X';// dead to alive
										}
}	

int main()
{
	int rows = 10;
	int cols = 10;
	int i,j, cIndex;
	char temp;
	
	// two sepreate array coalesced reads cuda
	char S[rows*cols];
	int C[rows*cols];
	ifstream fin;
	fin.open("./input.txt");
	
	for(i = 0; i < rows; i++)
	{
		for(j = 0; j<cols; j++)
		{   
			fin >> temp;
			cIndex = i*cols+j;
			S[cIndex]= temp;
			C[cIndex]= -1;
			
			//cout << A[i *cols + j].getStatus();
		}
		cout << endl;
	}
	
	char *A;
	int *B;
	hipMalloc(&A, rows*cols*(sizeof(char)));
	hipMalloc(&B, rows*cols*(sizeof(int)));//allocates bytes from device heap and returns pointer to allocated memory or null
	for(i = 0; i < rows; i++)
	{
			for(j = 0; j<cols; j++)
			{   
				hipMemcpy(&A[i*cols+j], &S[i*cols+j], sizeof(char), hipMemcpyHostToDevice);
				hipMemcpy(&B[i*cols+j], &C[i*cols+j], sizeof(int), hipMemcpyHostToDevice);
			}

				//	A[23].setCount(checkAdjCells(rows,cols,23, A));
				//	cout << "Status " << A[23].getStatus() << " Count" << A[23].getCount();
	}
	int l = 0;
	while(l< 10){
		        //     <<<number of blocks, number of threads per block>>>
	callCheck<<<1,100>>>(rows, cols, A, B);
	hipDeviceSynchronize();
	for(i = 0; i < rows; i++)
		{
				
				for(j = 0; j<cols; j++)
				{   
					
					hipMemcpy(&S[i*cols+j], &A[i*cols+j], sizeof(char), hipMemcpyDeviceToHost);
					hipMemcpy(&C[i*cols+j], &B[i*cols+j], sizeof(int), hipMemcpyDeviceToHost);
					//cout << i*cols+j << " index " << S[i*cols+j] << " status " << C[i*cols+j] << " count " << endl;
					cout << S[i*cols+j];
				}
				cout << endl;
					//	A[23].setCount(checkAdjCells(rows,cols,23, A));
					//	cout << "Status " << A[23].getStatus() << " Count" << A[23].getCount();
		}
	cout << endl;
	cout << endl;
	l++;
	}
	hipFree(A);
	hipFree(B);
	
	cin.get();
	//cudaMallocManaged(sizeof(char)*rows*cols);
	//cudaMemcpy(hostToDevice)

}


/*struct Cell{
	char status;
	int count;
};


int checkAdjCells(int rows, int cols, int cIndex, char A[])
	{
		int i, j, iIndex, jIndex, count = 0;
		iIndex = cIndex/cols; //row index
		jIndex = cIndex%cols; // col index
		for(i = iIndex-1; i <= iIndex+1; i++)
		{
			for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
			{
				// i<0 can't have negative index
				//i>rows j > cols can't have index larger than array Max
				if(i>=0 && i<=rows && j >=0 && j<= cols && A[i*cols+j] == 'X' && (i*cols+j!= cIndex)) count++;
			}
		}
		return count;
	}

char setStatus(int num, char status) //Each line ends with newline character \n (Unix formatting)
{
		//cout << num << " ";
		char newStat = status;
		if(status == 'X') //check if it's alive
		{
		if(num < 2) newStat = '-';//dead less than 2 living neighbours
		else if(num <= 3) newStat = 'X'; //do nothing status is already alive
		else newStat = '-';//dead greater than 3 living neighbours
		}
		else{
			if(num == 3) newStat = 'X'; // dead to alive
		}
		return newStat;
}

char getStatus()
{
	return status;
}
int getCount()
{
	return count;
}
void setCount(int num)
{8
		count = num;
}

void initStatus(char c)
{
	status = c;
}

*/
/*
char setStatus(int num, char status) //Each line ends with newline character \n (Unix formatting)
{
		char newStat = status;
		if(status == 'X') //check if it's alive
		{
		if(num < 2) newStat = '-';//dead less than 2 living neighbours
		else if(num <= 3) cout << "ok"; //do nothing status is already alive
		else newStat = '-';//dead greater than 3 living neighbours
		}
		else{
			if(num == 3) newStat = 'X'; // dead to alive
		}
		return newStat;
}


void checkAdjCells(int rows, int cols, int cIndex, Cell A[])
	{
		int i, j, iIndex, jIndex, count = 0;
		iIndex = cIndex/cols; //row index
		jIndex = cIndex%cols; // col index
		for(i = iIndex-1; i <= iIndex+1; i++)
		{
			for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
			{
				// i<0 can't have negative index
				//i>rows j > cols can't have index larger than array Max
				if(i>=0 && i<=rows && j >=0 && j<= cols && A[i*cols+j].status == 'X' && (i*cols+j!= cIndex)) count++;
			}
		}
		A[cIndex].count = count;
	}

for(int m = 0; m < 5; m++){
	for(i = 0; i < rows; i++)
		{
			for(j = 0; j<cols; j++)
			{   
				cIndex = i*cols+j;
				C[cIndex]= checkAdjCells(rows,cols,cIndex, S);
				cout << S[cIndex] << C[cIndex];
				//cout << A[i *cols + j].getStatus();
			}
			cout << endl;
		}
	cout << endl;
	for(i = 0; i < rows; i++)
			{
				for(j = 0; j<cols; j++)
				{   
					cIndex = i*cols+j;
					S[cIndex] = setStatus(C[cIndex], S[cIndex]);
					//cout << A[i *cols + j].getStatus();
					cout << S[cIndex] << C[cIndex];
				}
				cout << endl;
			}
}
	
	cout << endl;
	cout << endl;
	fin.close();
*/
