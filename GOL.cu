
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <stdlib.h>
#include <string>

using namespace std;

const int THREADS_PER_BLOCK = 512;

__global__
void callCheck(int rows, int cols,char A[], int B[])
{
	int i, k, j, count, iIndex, jIndex;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    int rowIndex, colIndex;//index of current thread
	//int stride = blockDim.x *gridDim.x; total threads grid striping 
				//checkAdjCells(rows,cols, k, A);
				iIndex = i/cols; //row index
				jIndex = i%cols; // col index
				count = 0;
				for(k = iIndex-1; k <= iIndex+1; k++)
				{
					for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
					{
						// i<0 can't have negative index
						//i>rows j > cols can't have index larger than array Max
						if(k<0) rowIndex = rows-1;
						else if(k>=rows)rowIndex = 0;
						else rowIndex = k;
						if(j<0) colIndex = cols-1;
						else if(j>=cols) colIndex = 0;
						else colIndex = j;
						if (A[rowIndex*cols+colIndex] == 'X' && (rowIndex*cols+colIndex!= i)) count++;
					}
				}
									    B[i] = count;
										if(A[i] == 'X') //check if it's alive
										{
										if(B[i] < 2) A[i] = '-';//dead less than 2 living neighbours
										else if(B[i] <= 3) A[i] = 'X'; //do nothing status is already alive
										else A[i] = '-';//dead greater than 3 living neighbours
										}
										else{ //dead cell
											if(B[i] == 3) A[i] = 'X';// dead to alive
										}
}	

int main(int argc, char *argv[])
{
	int i,j, rows, cols;
	char temp = '=';
	rows = 1;
	cols = 1;
	// two sepreate array coalesced reads cudachar S[rows*cols];
	vector<char> tempS;
	ifstream fin;
	ofstream fout;
	bool printAll = false;
		int opts = 0;
		string input;
		int iterations = 1;
		while(opts < argc)
		{
			if(string(argv[opts]) == "-i") iterations = strtol(argv[opts+1], NULL, 10);
			if(string(argv[opts]) == "-v") printAll = true;
			if(opts == argc-1){
				string ext;
				string temp = argv[opts];
				for(i = temp.length()-4; i < temp.length(); i++) ext += temp[i];
				if(ext == ".txt") input = temp;
			}
			opts++;
		}
	fin.open(input.c_str());
	if(fin){
	fout.open("output.txt");
	i=0;
	fin >> temp;
	int totalcount = 0; //total number of elements
	while(!fin.eof())
	{
		totalcount++;
		if(temp == 'X' || temp == '-')
		{
			if(fin.peek() == '\n')
				{
				rows++;
				}
			else if(rows == 1)cols++;
			tempS.push_back(temp); //read in status 
		}else cout << "Invalid input = " << temp << endl;
		fin >> temp;
		i++;
	}
	fin.close();
	if(cols*rows >8){
		if(totalcount== rows*cols){
	int C[rows*cols];
	char S[rows*cols];
	for(j=0; j<rows*cols; j++)
	{
		C[j]=-1;
		S[j]= tempS[j];
		
	}
	
	tempS.clear();
	
	fout << "Initial step" << endl;
	for(i = 0; i < rows; i++)
			{
					
					for(j = 0; j<cols; j++)
					{   
						
						
						fout << S[i*cols+j];
					}
					fout << endl;
			}
	fout << endl;
		fout << endl;
	char *A;
	int *B;
	hipMalloc((void** ) &A, rows*cols*(sizeof(char)));
	hipMalloc((void** ) &B, rows*cols*(sizeof(int)));//allocates bytes from device heap and returns pointer to allocated memory or null
	hipMemcpy(A, S, rows*cols*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(B, C, rows*cols*sizeof(int), hipMemcpyHostToDevice);
	
	int l = 0;
	while(l < iterations){
 //     <<<number of blocks, number of threads per block>>>
	if(rows*cols < THREADS_PER_BLOCK)callCheck<<<1,rows*cols>>>(rows, cols, A, B); // one block of rows*cols threads
	else callCheck<<<cols*rows/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(rows,cols,A,B);
	hipDeviceSynchronize();
	hipMemcpy(S, A, rows*cols*sizeof(char), hipMemcpyDeviceToHost);
	hipMemcpy(C, B, rows*cols*sizeof(int), hipMemcpyDeviceToHost);
	if(printAll == true || l == iterations-1)
	{
		fout << "Step " << l+1 << endl;
	for(i = 0; i < rows; i++)
		{
				
				for(j = 0; j<cols; j++)
				{   
					fout << S[i*cols+j];
				}
				fout << endl;
		}
	fout << endl;
	fout << endl;
	}
	l++;
	}
	hipFree(A);
	hipFree(B);
	fout.close();
	cout << "All Done";
		}else cout << "Matrix is not even";
	}else cout <<"Matrix must be at least 9 elements";
	}else cout<< "Could not find the input file please try running again with valid file";
	cin.get();

}


/*struct Cell{
	char status;
	int count;
};


int checkAdjCells(int rows, int cols, int cIndex, char A[])
	{
		int i, j, iIndex, jIndex, count = 0;
		iIndex = cIndex/cols; //row index
		jIndex = cIndex%cols; // col index
		for(i = iIndex-1; i <= iIndex+1; i++)
		{
			for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
			{
				// i<0 can't have negative index
				//i>rows j > cols can't have index larger than array Max
				if(i>=0 && i<=rows && j >=0 && j<= cols && A[i*cols+j] == 'X' && (i*cols+j!= cIndex)) count++;
			}
		}
		return count;
	}

char setStatus(int num, char status) //Each line ends with newline character \n (Unix formatting)
{
		//cout << num << " ";
		char newStat = status;
		if(status == 'X') //check if it's alive
		{
		if(num < 2) newStat = '-';//dead less than 2 living neighbours
		else if(num <= 3) newStat = 'X'; //do nothing status is already alive
		else newStat = '-';//dead greater than 3 living neighbours
		}
		else{
			if(num == 3) newStat = 'X'; // dead to alive
		}
		return newStat;
}

char getStatus()
{
	return status;
}
int getCount()
{
	return count;
}
void setCount(int num)
{8
		count = num;
}

void initStatus(char c)
{
	status = c;
}

*/
/*
char setStatus(int num, char status) //Each line ends with newline character \n (Unix formatting)
{
		char newStat = status;
		if(status == 'X') //check if it's alive
		{
		if(num < 2) newStat = '-';//dead less than 2 living neighbours
		else if(num <= 3) cout << "ok"; //do nothing status is already alive
		else newStat = '-';//dead greater than 3 living neighbours
		}
		else{
			if(num == 3) newStat = 'X'; // dead to alive
		}
		return newStat;
}


void checkAdjCells(int rows, int cols, int cIndex, Cell A[])
	{
		int i, j, iIndex, jIndex, count = 0;
		iIndex = cIndex/cols; //row index
		jIndex = cIndex%cols; // col index
		for(i = iIndex-1; i <= iIndex+1; i++)
		{
			for (j = jIndex-1; j <= jIndex+1; j++) //Each line ends with newline character \n (Unix formatting)
			{
				// i<0 can't have negative index
				//i>rows j > cols can't have index larger than array Max
				if(i>=0 && i<=rows && j >=0 && j<= cols && A[i*cols+j].status == 'X' && (i*cols+j!= cIndex)) count++;
			}
		}
		A[cIndex].count = count;
	}

for(int m = 0; m < 5; m++){
	for(i = 0; i < rows; i++)
		{
			for(j = 0; j<cols; j++)
			{   
				cIndex = i*cols+j;
				C[cIndex]= checkAdjCells(rows,cols,cIndex, S);
				cout << S[cIndex] << C[cIndex];
				//cout << A[i *cols + j].getStatus();
			}
			cout << endl;
		}
	cout << endl;
	for(i = 0; i < rows; i++)
			{
				for(j = 0; j<cols; j++)
				{   
					cIndex = i*cols+j;
					S[cIndex] = setStatus(C[cIndex], S[cIndex]);
					//cout << A[i *cols + j].getStatus();
					cout << S[cIndex] << C[cIndex];
				}
				cout << endl;
			}
}
	
	cout << endl;
	cout << endl;
	fin.close();
*/
